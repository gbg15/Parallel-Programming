/*
Prime sieve code for CS 4380 / CS 5351

Copyright (c) 2017, Texas State University. All rights reserved.

Redistribution in source or binary form, with or without modification,
is not permitted. Use in source and binary forms, with or without
modification, is only permitted for academic use in CS 4380 or CS 5351
at Texas State University.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

Author: Martin Burtscher
*/

#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <sys/time.h>

static const int ThreadsPerBlock = 256;

static __global__ void SieveInitKernel(const long top, bool array[])
{
  const int idx = threadIdx.x + blockIdx.x * blockDim.x; //todo: initialize array here
  if(idx < top)
    array[idx] = true;
}

static __global__ void SieveComputeKernel(const long top, bool array[])
{
  const int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if ((idx >= 2) && (idx * idx < top)) {
    long j = idx * idx;
    while (j < top){
      array[j] = false; //todo: remove multiples here
      j += idx;
    }
  }
}

static bool isPrime(long val)
{
  if (val < 2) return false;
  for (long i = 2; i * i <= val; i++) {
    if ((val % i) == 0) return false;
  }
  return true;
}

static void CheckCuda()
{
  hipError_t e;
  hipDeviceSynchronize();
  if (hipSuccess != (e = hipGetLastError())) {
    fprintf(stderr, "CUDA error %d: %s\n", e, hipGetErrorString(e));
    exit(-1);
  }
}

int main(int argc, char *argv[])
{
  printf("Prime Sieve v1.0 [CUDA]\n");

  // check command line
  if (argc != 2) {fprintf(stderr, "usage: %s maximum\n", argv[0]); exit(-1);}
  const long top = atol(argv[1]);
  if (top < 23) {fprintf(stderr, "error: maximum must be at least 23\n"); exit(-1);}
  printf("computing prime numbers up to but not including %ld\n", top);

  // allocate array
  bool* array = new bool[top];
  bool* array_d;
  //todo: allocate array_d here
  if(hipSuccess != hipMalloc((void**)&array_d, (top * sizeof(bool)))) {fprintf(stderr, "memory failed to allocate\n"); exit(-1);}

  // start time
  timeval start, end;
  gettimeofday(&start, NULL);

  // call kernel
  SieveInitKernel<<<(top + ThreadsPerBlock - 1) / ThreadsPerBlock, ThreadsPerBlock>>>(top, array_d);
  CheckCuda();
  const int sqrt_top = sqrt(top);
  SieveComputeKernel<<<(sqrt_top + ThreadsPerBlock - 1) / ThreadsPerBlock, ThreadsPerBlock>>>(top, array_d);
  CheckCuda();
  if(hipSuccess != hipMemcpy(array, array_d, top * sizeof(bool), hipMemcpyDeviceToHost)) {fprintf(stderr, "copying from device failed\n"); exit(-1);}//todo: copy results back to CPU here

  // end time
  gettimeofday(&end, NULL);
  double runtime = end.tv_sec + end.tv_usec / 1000000.0 - start.tv_sec - start.tv_usec / 1000000.0;
  printf("compute time: %.4f s\n", runtime);

  // print part of result
  for (long i = 2; i < 10; i++) {
    if (array[i]) {
      printf(" %ld", i);
    }
  }
  printf(" ...");
  for (long i = top - 10; i < top; i++) {
    if (array[i]) {
      printf(" %ld", i);
    }
  }
  printf("\n");

  // verify result
  if (top < 10000000) {
    for (long i = 2; i < top; i++) {
      if (array[i] != isPrime(i)) {
        fprintf(stderr, "ERROR: wrong answer for %ld\n\n", i);
        exit(-1);
      }
    }
  }

  delete [] array;
  hipFree(array_d); //todo: free array_d here
  return 0;
}

